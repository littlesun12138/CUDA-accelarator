#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16


__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    // int c=Channel;
    // int sharememwidth=TILE_WIDTH+K-1;
    extern __shared__ float sharedmem[];
    // __shared__ float sharedmem[Channel][TILE_WIDTH+K-1][TILE_WIDTH+K-1];
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working
    int m = blockIdx.x;
    int b = blockIdx.z;
    int W_grid= (Width_out - 1)/ TILE_WIDTH  + 1;
    int h = (blockIdx.y / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_grid) * TILE_WIDTH + threadIdx.x;
    

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    //SHARED FILLED

    for(int i = 0; i < Channel; i++){
        // some thread need to add more than one value to sharedmem
        for(int j=threadIdx.y;j<TILE_WIDTH+K-1;j+=TILE_WIDTH){
            for(int k=threadIdx.x;k<TILE_WIDTH+K-1;k+=TILE_WIDTH){
                //sharedmem boundary check
                if((blockIdx.y / W_grid) * TILE_WIDTH + j<Height && (blockIdx.y % W_grid) * TILE_WIDTH + i<Width){
                    sharedmem[i*(TILE_WIDTH+K-1)*(TILE_WIDTH+K-1)+j*(TILE_WIDTH+K-1)+k]=in_4d(b,i,(blockIdx.y / W_grid) * TILE_WIDTH+j, (blockIdx.y % W_grid) * TILE_WIDTH+k);
                    //sharedmem[i][j][k]=in_4d(b,i,(blockIdx.y / W_grid) * TILE_WIDTH+j, (blockIdx.y % W_grid) * TILE_WIDTH+k);
                }
                else{
                    sharedmem[i*(TILE_WIDTH+K-1)*(TILE_WIDTH+K-1)+j*(TILE_WIDTH+K-1)+k]=0.0;
                    //sharedmem[i][j][k]=0.0;
                }
            }
        }
        
    }
    __syncthreads();    

    if(w < Width_out && h < Height_out){
        float acc = 0.0f;
        for(int i = 0; i < Channel; i++){
            for(int j = 0; j < K; j++){                    
                for(int k = 0; k < K; k++){
                    acc += sharedmem[i*(TILE_WIDTH+K-1)*(TILE_WIDTH+K-1)+(threadIdx.y+j)*(TILE_WIDTH+K-1)+threadIdx.x+k] * mask_4d(m, i , j , k); 
                    //acc += sharedmem[i][threadIdx.y+j][threadIdx.x+k] * mask_4d(m, i , j , k); 
                }
            }
        }
        out_4d(b, m, h, w) = acc;
    }
    // __syncthreads();    

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void **) device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpyToSymbol(HIP_SYMBOL(MASK), host_mask, Map_out * Channel * K * K * sizeof(float));
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(Map_out, ceil(1.0 * Height_out / TILE_WIDTH) * ceil(1.0 * Width_out / TILE_WIDTH), Batch);

    size_t smsize=sizeof(float) *Channel*(TILE_WIDTH+K-1)*(TILE_WIDTH+K-1);


    conv_forward_kernel<<<dimGrid,dimBlock,smsize>>>(device_output,device_input,device_mask,Batch,Map_out,Channel,Height,Width,K);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, Batch * Map_out * (Height-K+1) * (Width-K+1) * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}