#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    // int c=Channel;
    // int sharememwidth=TILE_WIDTH+K-1;
    // extern __shared__ float sharedmem[];

    __shared__ float maskmem[TILE_WIDTH*TILE_WIDTH];
    __shared__ float sharedmem[TILE_WIDTH*TILE_WIDTH];
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working
    int m = blockIdx.x*TILE_WIDTH+threadIdx.x;
    int b = blockIdx.z;
    int W_grid= (Width_out - 1)/ TILE_WIDTH  + 1;
    int h = (blockIdx.y* TILE_WIDTH + threadIdx.y)/ Width_out;
    int w = (blockIdx.y* TILE_WIDTH + threadIdx.y) % Width_out ;
    

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    //SHARED FILLED
    
    
    float sum=0.0;
    
    for (int i = 0; i < ceil((Channel * K * K)/(1.0*TILE_WIDTH)); i++){
        int seq=i * TILE_WIDTH;
        if((threadIdx.y+seq) < Channel * K*K  && m<Map_out){
            maskmem[threadIdx.x*TILE_WIDTH+threadIdx.y]=mask[m* (Channel * K * K) + seq + threadIdx.y];
        }
        else{
                maskmem[threadIdx.x*TILE_WIDTH+threadIdx.y]=0.0;
        }
        //boundary check for  output dim and mapout dim 
        if( ((seq + threadIdx.x) < Channel * K * K) && (blockIdx.y*TILE_WIDTH + threadIdx.y)<Height_out*Width_out){
            
            sharedmem[threadIdx.x*TILE_WIDTH+threadIdx.y]=input[b*(Height * Width*Channel)+ (seq+threadIdx.x)/(K*K)*(Height * Width) + (seq+threadIdx.x)%(K*K)/K*Width+(blockIdx.y* TILE_WIDTH + threadIdx.y)/Width_out*Width + (seq+threadIdx.x)%(K*K)%K+(blockIdx.y* TILE_WIDTH + threadIdx.y)%Width_out];
            
            
        }
        else{
            sharedmem[threadIdx.x*TILE_WIDTH+threadIdx.y]=0.0;
        }
        //this is used for both shared mem sync of mask and input
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            sum += maskmem[threadIdx.x*TILE_WIDTH+k] * sharedmem[k*TILE_WIDTH+threadIdx.y];
        }

        __syncthreads();
    }        
    if( (blockIdx.y*TILE_WIDTH + threadIdx.y)<Height_out*Width_out && m<Map_out){
        output[b* (Map_out * Height_out * Width_out) + m * (Height_out * Width_out) + (h*Width_out + w)]=sum;
    }




    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;           


    hipMalloc((void **) device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);


    // hipMemcpyToSymbol(HIP_SYMBOL(MASK), host_mask, Map_out * Channel * K * K * sizeof(float));
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    // int Height_out = Height - K + 1;
    // int Width_out = Width - K + 1;
    // float *unroll_input_ptr;  
    // hipMalloc((void **) &unroll_input_ptr, Batch * Channel *K*K* Height_out * Width_out * sizeof(float));    
    // dim3 dimBlock(TILE_WIDTH, 1, 1);
    // dim3 dimGrid( ceil( Height_out* Width_out*Channel / (1.0*TILE_WIDTH)),1, Batch);
    // unroll<<<dimGrid,dimBlock>>>(unroll_input_ptr,device_input,Channel, Height,Width, K);
    // hipDeviceSynchronize();

    // //shared mem can reuse input feature in different maps,so need to use mapout as one dimension
    // dim3 dimBlock1(TILE_WIDTH, TILE_WIDTH, 1);
    // dim3 dimGrid1( ceil(Width_out * Height_out /(1.0*TILE_WIDTH) ),ceil(Map_out/(1.0*TILE_WIDTH)) , Batch);

    // //shared mem need to be larger.
    // // size_t smsize=sizeof(float) *TILE_WIDTH*TILE_WIDTH;


    // // conv_forward_kernel<<<dimGrid1,dimBlock1,smsize>>>(device_output,unroll_input_ptr,device_mask,Batch,Map_out,Channel,Height,Width,K);
    // conv_forward_kernel<<<dimGrid1,dimBlock1>>>(device_output,unroll_input_ptr,device_mask,Batch,Map_out,Channel,Height,Width,K);
    // hipFree(unroll_input_ptr);
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    // hipMalloc((void **) &unroll_input_ptr, Batch * Channel *K*K* Height_out * Width_out * sizeof(float));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(ceil(Map_out/(1.0*TILE_WIDTH)), ceil(Width_out * Height_out /(1.0*TILE_WIDTH) ), Batch);

    // size_t shared_mem_size = TILE_WIDTH  * TILE_WIDTH  * sizeof(float);

    // Launch the fused kernel
    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
    // hipDeviceSynchronize();

    // // Check for errors
    // hipError_t error = hipGetLastError();
    // if (error != hipSuccess) {
    //     std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, Batch * Map_out * (Height-K+1) * (Width-K+1) * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
    
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}